#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include "kernel.h"

hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);

__global__ void addKernel(Coords* c, const Coords* a, const Coords* b, float t, int numParticulas)
{
    const float G = 0.00001f;
    vec2 sum = vec2(0, 0);
    int i = threadIdx.x;
    for (int j = 0; j < i; ++j) {
        float d = length(a[j] - a[i]);
        if (d >= 0.01f) sum += (G / (d * d * d)) * (a[j] - a[i]);
    }
    for (int j = i + 1; j < numParticulas; ++j) {
        float d = length(a[j] - a[i]);
        if (d >= 0.01f) sum += (G / (d * d * d)) * (a[j] - a[i]);
    }

    c[i] = a[i] + a[i] - b[i] + (t * t) * sum;

}

// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size)
{
    hipError_t cudaStatus;

    // Launch a kernel on the GPU with one thread for each element.
   // ------------------> USAR ESTO addKernel<<<1, size>>>(dev_c, dev_a, dev_b);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    // ------------------> USAR ESTO cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    
    return cudaStatus;
}

void Kernel::kernel()
{

}

void Kernel::CUDAliberar()
{
    hipError_t cudaStatus;
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
    }

}

void Kernel::CUDASimular(Coords* coordsPrevias, Coords* coords, Coords* coordsSiguientes, int numParticulas, float pasoT)
{
    Coords* dev_previas = 0;
    Coords* dev_coords = 0;
    Coords* dev_siguientes = 0;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_siguientes, numParticulas * sizeof(Coords));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_previas, numParticulas * sizeof(Coords));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_coords, numParticulas * sizeof(Coords));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_previas, coordsPrevias, numParticulas * sizeof(Coords), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_coords, coords, numParticulas * sizeof(Coords), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    addKernel <<<1, numParticulas >>> (dev_siguientes, dev_coords, dev_previas, pasoT, numParticulas);
    cudaStatus = hipMemcpy(coordsSiguientes, dev_siguientes, numParticulas * sizeof(Coords), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }


Error:
    hipFree(dev_siguientes);
    hipFree(dev_coords);
    hipFree(dev_previas);

}
